#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N (33 * 1024)

__global__ void add(long int *a, long int *b, long int *c)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   while (tid < N) {
       c[tid] = a[tid] + b[tid];
       // in the GPU implementation we consider the number of parallel threads
       // launched to be the number of processors, although the actual GPU may
       // have fewer processing units that this, we think of each thread as
       // logically executing in parallle and the allow the hardware to
       // scheudle the actual execution
       tid += blockDim.x * gridDim.x;
   }
}

int main(int argc, char *argv[])
{
    long int a[N], b[N], c[N];
    long int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void **) &dev_a, N * sizeof(long int)));
    HANDLE_ERROR(hipMalloc((void **) &dev_b, N * sizeof(long int)));
    HANDLE_ERROR(hipMalloc((void **) &dev_c, N * sizeof(long int)));

    // fill the arrays 'a' and 'b' on the CPU
    for (long int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(long int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(long int), hipMemcpyHostToDevice));

    add<<<128, 128>>>(dev_a, dev_b, dev_c);

    // copy the arrays 'c' back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(long int), hipMemcpyDeviceToHost));

    // display the results
    for (int i = 0; i < N; ++i) {
        printf("%ld + %ld = %ld\n", a[i], b[i], c[i]);
    }

    bool success = true;
    for (int i = 0; i < N; ++i) {
        if ((a[i] + b[i]) != c[i]) {
            printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if (success) {
        printf("We did it!\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
