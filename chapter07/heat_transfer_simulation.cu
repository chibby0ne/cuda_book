#include <iostream>
#include <hip/hip_runtime.h>

#define CUDA_CALL(call)     \
{                           \
cudaError_t result = call;  \
if (hipSuccess != result) {       \
    std::cerr << "CUDA error: " << result << " in " << __FILE__ << ":" << __LINE__ << " : " << hipGetErrorString(result) << " (" << call << ")" << std::endl; \
} \
}


__global__ void add(int a, int b, int *c) 
{
    *c = a + b;
}

int main(int argc, char *argv[])
{
    int c;
    int *dev_c;
    CUDA_CALL(hipMalloc((void **) &dev_c, sizeof(int)))

    add<<<1, 1>>>(2, 7, &c);

    CUDA_CALL(hipMemcpy(&c,
            dev_c,
            sizeof(int),
            hipMemcpyDeviceToHost));

    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);
    return 0;

}
